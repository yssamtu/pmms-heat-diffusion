#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
// #include <cstring>
#include "timer.h"

enum {
    image_height = 1024,
    image_width = 1024,
    filter_height = 5,
    filter_width = 5,
    border_height =  filter_height & -2,
    border_width = filter_width & -2,
    input_height = image_height + border_height,
    input_width = image_width + border_width,
    block_size_x = 32,
    block_size_y = 16,
    SEED = 1234
};

using std::isnan;
using std::fprintf;
using std::printf;
using std::puts;
using std::calloc;
using std::free;
using std::malloc;
using std::cout;
using std::endl;

void convolutionSeq(float *output, float *input, float *filter) {
    //for each pixel in the output image

  timer sequentialTime = timer("Sequential");
  
  sequentialTime.start();

    for (int y = 0; y < image_height; ++y) {
        for (int x = 0; x < image_width; ++x) { 
	        output[y * image_width + x] = 0.0f;
            //for each filter weight
            for (int i = 0; i < filter_height; ++i) {
                for (int j = 0; j < filter_width; ++j) {
                    output[y * image_width + x] += input[(y + i) * input_width + x + j] * filter[i * filter_width + j];
                }
            }
	        output[y * image_width + x] /= 35.0f;
        }
    }
  
  sequentialTime.stop(); 
  cout << "convolution (sequential): \t\t" << sequentialTime << endl;

}


__global__ void convolution_kernel_naive(float *output, float *input, float *filter) {

}

void convolutionCUDA(float *output, float *input, float *filter) {
    float *d_input; float *d_output; float *d_filter;
    hipError_t err;
    timer kernelTime = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // memory allocation
    err = hipMalloc((void **)&d_input, input_height * input_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMalloc d_input: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **)&d_output, image_height * image_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMalloc d_output: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **)&d_filter, filter_height * filter_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMalloc d_filter: %s\n", hipGetErrorString(err));

    memoryTime.start();
    // host to device 
    err = hipMemcpy(d_input, input, input_height * input_width * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMemcpy host to device input: %s\n", hipGetErrorString(err));
    err = hipMemcpy(d_filter, filter, filter_height * filter_width * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMemcpy host to device filter: %s\n", hipGetErrorString(err));
    
    // zero the result array 
    err = hipMemset(d_output, 0, image_height * image_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMemset output: %s\n", hipGetErrorString(err));
    memoryTime.stop();
    //setup the grid and thread blocks
    //thread block size
    dim3 threads(block_size_x, block_size_y);
    //problem size divided by thread block size rounded up
    dim3 grid(int(ceilf(image_width / (float)threads.x)), int(ceilf(image_height / (float)threads.y)));

    //measure the GPU function
    kernelTime.start();
    convolution_kernel_naive<<<grid, threads>>>(d_output, d_input, d_filter);
    hipDeviceSynchronize();
    kernelTime.stop();
 
    //check to see if all went well
    err = hipGetLastError();
    if (err != hipSuccess)
        fprintf(stderr, "Error during kernel launch convolution_kernel: %s\n", hipGetErrorString(err));

    //copy the result back to host memory
    memoryTime.start();
    err = hipMemcpy(output, d_output, image_height * image_width * sizeof(float), hipMemcpyDeviceToHost);
    memoryTime.stop();
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMemcpy device to host output: %s\n", hipGetErrorString(err));
 
    err = hipFree(d_input);
    if (err != hipSuccess)
        fprintf(stderr, "Error in freeing d_input: %s\n", hipGetErrorString(err));
    err = hipFree(d_output);
    if (err != hipSuccess)
        fprintf(stderr, "Error in freeing d_output: %s\n", hipGetErrorString(err));
    err = hipFree(d_filter);
    if (err != hipSuccess)
        fprintf(stderr, "Error in freeing d_filter: %s\n", hipGetErrorString(err));

    cout << "convolution (kernel): \t\t" << kernelTime << endl;
    cout << "convolution (memory): \t\t" << memoryTime << endl;

}

int compare_arrays(float *a1, float *a2, int n) {
    int errors = 0;
    int print = 0;

    for (int i = 0; i < n; ++i) {

        if (isnan(a1[i]) || isnan(a2[i])) {
            ++errors;
            if (print < 10) {
                ++print;
                fprintf(stderr, "Error NaN detected at i=%d,\t a1= %10.7e \t a2= \t %10.7e\n", i, a1[i], a2[i]);
            }
        }

        float diff = (a1[i] - a2[i]) / a1[i];
        if (diff > 1e-6f) {
            ++errors;
            if (print < 10) {
                ++print;
                fprintf(stderr, "Error detected at i=%d, \t a1= \t %10.7e \t a2= \t %10.7e \t rel_error=\t %10.7e\n", i, a1[i], a2[i], diff);
            }
        }

    }

    return errors;
}
        

int main()
{
    //allocate arrays and fill them
    float *input = (float *)malloc(input_height * input_width * sizeof(float));
    float *output1 = (float *)calloc(image_height * image_width, sizeof(float));
    float *output2 = (float *)calloc(image_height * image_width, sizeof(float));
    float *filter = (float *)malloc(filter_height * filter_width * sizeof(float));

    for (int i = 0; i < input_height * input_width; ++i) {
        input[i] = (float)(i % SEED);
    }

//THis is specific for a W==H smoothening filteri, where W and H are odd.
    for (int i = 0; i < filter_height * filter_width; ++i) { 
        filter[i] = 1.0f;
    }

    for (int i = filter_width + 1; i < (filter_height - 1) * filter_width; ++i) {
	    if (i % filter_width > 0 && i % filter_width < filter_width - 1)
            filter[i] += 1.0f; 
    }

    filter[filter_width * filter_height >> 1] = 3.0f;

    for (int i = 0; i < filter_height; ++i) {
        for (int j = 0; j < filter_width; ++j) {
            printf("%f ", filter[i * filter_width + j]);
        }
        puts("");
    }
//end initialization
   
    // //measure the CPU function
    // convolutionSeq(output1, input, filter);
    // //measure the GPU function
    // convolutionCUDA(output2, input, filter);


    // //check the result
    // int errors += compare_arrays(output1, output2, image_height * image_width);
    // if (errors > 0)
    //     printf("TEST FAILED! %d errors!\n", errors);
    // else
    //     puts("TEST PASSED!");


    free(filter);
    free(input);
    free(output1);
    free(output2);

    return 0;
}


