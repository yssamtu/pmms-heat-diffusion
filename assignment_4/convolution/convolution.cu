#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
// #include <cstring>
#include "timer.h"

enum {
    image_height = 1024,
    image_width = 1024,
    filter_height = 5,
    filter_width = 5,
    border_height =  filter_height & -2,
    border_width = filter_width & -2,
    input_height = image_height + border_height,
    input_width = image_width + border_width,
    block_size_x = 32,
    block_size_y = 16,
    SEED = 1234
};

using std::isnan;
using std::fprintf;
using std::printf;
using std::puts;
using std::calloc;
using std::free;
using std::malloc;
using std::cout;
using std::endl;

static void convolutionSeq(float *output, float *input, float *filter);
__global__ void convolution_kernel_naive(float *output, float *input, float *filter);
static void convolutionCUDA(float *output, float *input, float *filter);
static int compare_arrays(float *a1, float *a2, int n);

int main()
{
    // Allocate arrays and fill them
    float *input = static_cast<float *>(malloc(input_height * input_width * sizeof(float)));
    float *output1 = static_cast<float *>(calloc(image_height * image_width, sizeof(float)));
    float *output2 = static_cast<float *>(calloc(image_height * image_width, sizeof(float)));
    float *filter = static_cast<float *>(malloc(filter_height * filter_width * sizeof(float)));

    for (int i = 0; i < input_height * input_width; ++i) {
        input[i] = static_cast<float>(i % SEED);
    }

// This is specific for a W == H smoothening filter i, where W and H are odd.
    for (int i = 0; i < filter_height * filter_width; ++i) {
        filter[i] = 1.0f;
    }

    for (int i = filter_width + 1; i < (filter_height - 1) * filter_width; ++i) {
	    if (i % filter_width > 0 && i % filter_width < filter_width - 1)
            filter[i] += 1.0f;
    }

    filter[filter_width * filter_height >> 1] = 3.0f;

// End initialisation

    // Measure the CPU function
    convolutionSeq(output1, input, filter);
    // Measure the GPU function
    // convolutionCUDA(output2, input, filter);


    // // Check the result
    // int errors += compare_arrays(output1, output2, image_height * image_width);
    // if (errors > 0)
    //     printf("TEST FAILED! %d errors!\n", errors);
    // else
    //     puts("TEST PASSED!");


    free(input);
    free(output1);
    free(output2);
    free(filter);

    return 0;
}

static void convolutionSeq(float *output, float *input, float *filter)
{
    // For each pixel in the output image

  timer sequentialTime = timer("Sequential");

  sequentialTime.start();

    for (int y = 0; y < image_height; ++y) {
        for (int x = 0; x < image_width; ++x) {
	        output[y * image_width + x] = 0.0f;
            // For each filter weight
            for (int i = 0; i < filter_height; ++i) {
                for (int j = 0; j < filter_width; ++j) {
                    output[y * image_width + x] += input[(y + i) * input_width + x + j] * filter[i * filter_width + j];
                }
            }
	        output[y * image_width + x] /= 35.0f;
        }
    }

  sequentialTime.stop();
  cout << "convolution (sequential): \t\t" << sequentialTime << endl;

}

__global__ void convolution_kernel_naive(float *output, float *input, float *filter)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
}

static void convolutionCUDA(float *output, float *input, float *filter)
{
    // Memory allocation
    float *d_input = nullptr;
    hipError_t err = hipMalloc(&d_input, input_height * input_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMalloc d_input: %s\n", hipGetErrorString(err));
    float *d_output = nullptr;
    err = hipMalloc(&d_output, image_height * image_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMalloc d_output: %s\n", hipGetErrorString(err));
    float *d_filter = nullptr;
    err = hipMalloc(&d_filter, filter_height * filter_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMalloc d_filter: %s\n", hipGetErrorString(err));

    timer memoryTime = timer("memoryTime");
    memoryTime.start();
    // Host to device
    err = hipMemcpy(d_input, input, input_height * input_width * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMemcpy host to device input: %s\n", hipGetErrorString(err));
    err = hipMemcpy(d_filter, filter, filter_height * filter_width * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMemcpy host to device filter: %s\n", hipGetErrorString(err));

    // Zero the result array
    err = hipMemset(d_output, 0, image_height * image_width * sizeof(float));
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMemset output: %s\n", hipGetErrorString(err));
    memoryTime.stop();
    // Set up the grid and thread blocks
    // Thread block size
    dim3 threads(block_size_x, block_size_y);
    // Problem size divided by thread block size rounded up
    dim3 grid(static_cast<unsigned>(ceilf(image_width / static_cast<float>(threads.x))), static_cast<unsigned>(ceilf(image_height / static_cast<float>(threads.y))));

    // Measure the GPU function
    timer kernelTime = timer("kernelTime");
    kernelTime.start();
    convolution_kernel_naive<<<grid, threads>>>(d_output, d_input, d_filter);
    hipDeviceSynchronize();
    kernelTime.stop();

    // Check to see if all went well
    err = hipGetLastError();
    if (err != hipSuccess)
        fprintf(stderr, "Error during kernel launch convolution_kernel: %s\n", hipGetErrorString(err));

    // Copy the result back to host memory
    memoryTime.start();
    err = hipMemcpy(output, d_output, image_height * image_width * sizeof(float), hipMemcpyDeviceToHost);
    memoryTime.stop();
    if (err != hipSuccess)
        fprintf(stderr, "Error in hipMemcpy device to host output: %s\n", hipGetErrorString(err));

    err = hipFree(d_input);
    if (err != hipSuccess)
        fprintf(stderr, "Error in freeing d_input: %s\n", hipGetErrorString(err));
    err = hipFree(d_output);
    if (err != hipSuccess)
        fprintf(stderr, "Error in freeing d_output: %s\n", hipGetErrorString(err));
    err = hipFree(d_filter);
    if (err != hipSuccess)
        fprintf(stderr, "Error in freeing d_filter: %s\n", hipGetErrorString(err));

    cout << "convolution (kernel): \t\t" << kernelTime << endl;
    cout << "convolution (memory): \t\t" << memoryTime << endl;

}

static int compare_arrays(float *a1, float *a2, int n)
{
    int errors = 0;
    int print = 0;

    for (int i = 0; i < n; ++i) {

        if (isnan(a1[i]) || isnan(a2[i])) {
            ++errors;
            if (print < 10) {
                ++print;
                fprintf(stderr, "Error NaN detected at i=%d,\t a1= %10.7e \t a2= \t %10.7e\n", i, a1[i], a2[i]);
            }
        }

        float diff = (a1[i] - a2[i]) / a1[i];
        if (diff > 1e-6f) {
            ++errors;
            if (print < 10) {
                ++print;
                fprintf(stderr, "Error detected at i=%d, \t a1= \t %10.7e \t a2= \t %10.7e \t rel_error=\t %10.7e\n", i, a1[i], a2[i], diff);
            }
        }

    }

    return errors;
}
